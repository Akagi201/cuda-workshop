// Add with a single thread on GPU

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c) { *c = a + b; }

int main() {
  int c;      // host copies
  int *dev_c; // device copies
  int size = sizeof(int);

  // Allocate space on device
  hipMalloc((void **)&dev_c, size);

  // Launch add() kernel on GPU
  add<<<1, 1>>>(8, 2, dev_c);

  // Copy result back to host
  hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);

  printf("%d\n", c);

  // Cleanup
  hipFree(dev_c);

  return 0;
}
