
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void hello_cuda_from_gpu() {
    printf("GPU: Hello, CUDA!\n");
}

void hello_cuda_from_cpu() {
    printf("CPU: Hello, CUDA!\n");
}

int main() {
    {
        // GPU设备端核函数所指定的总线程数 = gridDim * blockDim
        const int GRID_DIM = 2;          // grid网格大小（线程块数量）
        const int BLOCK_DIM = 8;         // block线程块大小（每个线程块中的线程数量）
        hello_cuda_from_gpu<<<GRID_DIM, BLOCK_DIM>>>();  // GPU设备端核函数调用
        hipDeviceSynchronize();        // 同步CPU主机端和GPU设备端
    }
    printf("\n");
    {
        hello_cuda_from_cpu();
    }
    return 0;
}